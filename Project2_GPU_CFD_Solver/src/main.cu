// Main CUDA source file for the GPU CFD Solver
/* Basic 2D GPU CFD Solver using CUDA */
/* Solves 2D incompressible Navier-Stokes using finite difference */

#include <stdio.h>
#include <hip/hip_runtime.h>

#define NX 128
#define NY 128
#define NSTEPS 1000

__global__ void update_velocity(float *u, float *v, float *u_new, float *v_new, float dt, float dx, float dy) {
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1;

    if (i < NX - 1 && j < NY - 1) {
        int idx = j * NX + i;
        u_new[idx] = u[idx] + dt * (u[idx - 1] - 2 * u[idx] + u[idx + 1]) / (dx * dx);
        v_new[idx] = v[idx] + dt * (v[idx - NX] - 2 * v[idx] + v[idx + NX]) / (dy * dy);
    }
}

int main() {
    size_t size = NX * NY * sizeof(float);
    float *u, *v, *u_new, *v_new;

    hipMallocManaged(&u, size);
    hipMallocManaged(&v, size);
    hipMallocManaged(&u_new, size);
    hipMallocManaged(&v_new, size);

    for (int i = 0; i < NX * NY; ++i) {
        u[i] = 0.0f;
        v[i] = 0.0f;
    }

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((NX + 15) / 16, (NY + 15) / 16);

    for (int t = 0; t < NSTEPS; ++t) {
        update_velocity<<<numBlocks, threadsPerBlock>>>(u, v, u_new, v_new, 0.01f, 0.01f, 0.01f);
        hipDeviceSynchronize();

        float *tmp_u = u; u = u_new; u_new = tmp_u;
        float *tmp_v = v; v = v_new; v_new = tmp_v;
    }

    hipFree(u); hipFree(v); hipFree(u_new); hipFree(v_new);
    return 0;
}
